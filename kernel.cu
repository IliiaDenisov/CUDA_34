#include "hip/hip_runtime.h"
﻿#include "Functions.h"

__device__ double derivativeFunction(double x, double y, double a)
{
    return sin((x + y) / a);
}

__global__ void solveDifferentianEquationGPU_kernel(unsigned char* img, size_t rows, size_t cols, unsigned char a, unsigned char b)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < (rows * cols * 3)) {
        img[id] = static_cast<unsigned char>((img[id] - a) * 255 / (b - a));
    }
}

void solveDifferentianEquationGPU(int BLK, int TPB, unsigned char* img, size_t rows, size_t cols, unsigned char a, unsigned char b)
{
    solveDifferentianEquationGPU_kernel << <BLK, TPB >> > (img, rows, cols, a, b);
}